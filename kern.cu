#include "hip/hip_runtime.h"
#include <math.h>
#include "nbody.h"

extern Z n;

V *dev_r, *dev_v;

static __global__ void kick(V *v, V *r, R dt, Z n)
{
  Z i = blockIdx.x * blockDim.x + threadIdx.x;
  Z j;

  if(i < n) {
    V dt_a = {0.0, 0.0, 0.0};

    for(j = 0; j < n; ++j) {
      V dr     = {r[i].x - r[j].x, r[i].y - r[j].y, r[i].z - r[j].z};
      R rr     = dr.x * dr.x + dr.y * dr.y + dr.z * dr.z + SOFTENING2;
      R dt_rrr = dt / (rr * sqrt(rr));

      dt_a.x -= dr.x * dt_rrr;
      dt_a.y -= dr.y * dt_rrr;
      dt_a.z -= dr.z * dt_rrr;
    }

    v[i].x += dt_a.x;
    v[i].y += dt_a.y;
    v[i].z += dt_a.z;
  }
}

static __global__ void drift(V *r, V *v, R dt, Z n)
{
  Z i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i < n) {
    r[i].x += v[i].x * dt;
    r[i].y += v[i].y * dt;
    r[i].z += v[i].z * dt;
  }
}

void evol(int ns, double dt)
{
  const int block_sz = 256;
  const int grid_sz = (n + block_sz - 1) / block_sz;

  R kdt = dt / 2; /* the first kick is a half step */
  Z i;
  for(i = 0; i < ns; ++i) {
    kick <<<grid_sz, block_sz>>>(dev_v, dev_r, kdt, n);
    drift<<<grid_sz, block_sz>>>(dev_r, dev_v,  dt, n);
    kdt = dt; /* all other kicks are full steps */
  }
  /* Last half-step correction */
  kick<<<grid_sz, block_sz>>>(dev_v, dev_r, dt / 2, n);
}
