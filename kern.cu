#include "hip/hip_runtime.h"
#include <math.h>
#include "nbody.h"

extern Z n;
extern V *r, *v;

V *dev_r, *dev_v;

static inline void kick(double dt)
{
  Z i, j;

  for(i = 0; i < n; ++i) {
    V dt_a = {0.0, 0.0, 0.0};

    for(j = 0; j < n; ++j) if(i != j) {
      V dr     = {r[i].x - r[j].x, r[i].y - r[j].y, r[i].z - r[j].z};
      R rr     = dr.x * dr.x + dr.y * dr.y + dr.z * dr.z + SOFTENING2;
      R dt_rrr = dt / (rr * sqrt(rr));

      dt_a.x -= dr.x * dt_rrr;
      dt_a.y -= dr.y * dt_rrr;
      dt_a.z -= dr.z * dt_rrr;
    }

    v[i].x += dt_a.x;
    v[i].y += dt_a.y;
    v[i].z += dt_a.z;
  }
}

static inline void drift(R dt)
{
  Z i;

  for(i = 0; i < n; ++i) {
    r[i].x += v[i].x * dt;
    r[i].y += v[i].y * dt;
    r[i].z += v[i].z * dt;
  }
}

void evol(int n, double dt)
{
  R kdt = dt / 2; /* the first kick is a half step */
  Z i;
  for(i = 0; i < n; ++i) {
    kick(kdt);
    drift(dt);
    kdt = dt; /* all other kicks are full steps */
  }
  /* Last half-step correction */
  kick(dt / 2);
}
